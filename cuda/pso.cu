#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <iomanip>
#include <limits>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

#define INF (1e38f*1e38f)

#define SEED 42
#define DIMENSION 2
#define NUMPARTICLES 256
#define NTHREADS 512

using namespace std;

__constant__ float pso_lower_bound;
__constant__ float pso_upper_bound;
__constant__ float pso_inertial_weight;
__constant__ float pso_cognitive_param;
__constant__ float pso_social_param;


__global__ void update_global_best(float *fitness, float *global_best_fit, int n_pop) {

    int size = n_pop;
    extern __shared__ float min_values[];

    int tid = threadIdx.x;
    if (tid < size) {
        min_values[tid] = fitness[tid];
    } else {
        min_values[tid] = INF;
    }
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            min_values[tid] = min_values[tid] < min_values[tid + stride] ? min_values[tid] : min_values[tid + stride];   //min(min_values[tid], min_values[tid + stride]);
        }
        __syncthreads();
    }

    if (tid == 0) {
        *global_best_fit = min_values[0] < *global_best_fit ? min_values[0] : *global_best_fit;
    }
}

__global__ void calc_fitness(
    float *population,
    float *fitness,
    int n_pop,
    int dimension
){

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int particle;
    int size = n_pop * dimension;

    while (i < size){
        float val = population[i];
        particle = i / dimension;
        atomicAdd(&fitness[particle], (val-2.0)*(val-2.0));
        i+=stride;
    }
}


__global__ void update_best_positions(
    float *population,
    float *best_pos,
    float *fitness,
    float *best_fit,
    int n_pop,
    int dimension
){

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int particle;
    int size = n_pop * dimension;

    while (i < size){
        particle = i / dimension;
        best_pos[i] = fitness[particle] < best_fit[particle] ? population[i] : best_pos[i];
        i+=stride;
    }
}

__global__ void update_best_fitness(
    float *fitness,
    float *best_fit,
    int n_pop
){

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (i < n_pop){
        best_fit[i] = fitness[i] < best_fit[i] ? fitness[i] : best_fit[i];
        i+=stride;
    }
}

__global__ void init_population(float *population, float *best_pos, int seed, int popsize, int dimension) {

    int total_dimension = popsize * dimension;
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (i < total_dimension) {
        hiprandState_t state;
        hiprand_init(seed, i, 0, &state);

        float random_value = hiprand_uniform(&state);
        population[i] = (pso_upper_bound - pso_lower_bound)*random_value + pso_lower_bound;
        i += stride;
    }
}

__global__ void update_positions(
    float *population,
    float *velocity,
    float *best_pos,
    float *global_best,
    int seed,
    int n_pop,
    int dimension) {

    int total_dimension = n_pop * dimension;
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int dim;

    while (i < total_dimension){
        hiprandState_t state;
        hiprand_init(seed, i, 0, &state);

        float r1 = hiprand_uniform(&state);
        float r2 = hiprand_uniform(&state);

        dim = i % dimension;

        velocity[i] = (pso_inertial_weight*velocity[i]) + \
                        (pso_cognitive_param*r1*(best_pos[i] - population[i])) + \
                        (pso_social_param*r2*(global_best[dim] - population[i]));

        population[i] = population[i] + velocity[i];
        population[i] = population[i] < pso_lower_bound ? pso_lower_bound : population[i];
        population[i] = population[i] > pso_upper_bound ? pso_upper_bound : population[i];

        i += stride;
    }
}

void print_population(float* dev_population, int pop_size) {
    float *h_population = (float*)malloc(pop_size * sizeof(float));
    hipMemcpy(
        h_population,
        dev_population,
        pop_size * sizeof(float),
        hipMemcpyDeviceToHost
    );
    for (int i = 0; i < pop_size; i++) {
        printf("population[%d] = %f\n", i, h_population[i]);
    }
    free(h_population);
}

void print_fitness(float* dev_fitness, int n_pop){
    float *h_fitness = (float*)malloc(n_pop * sizeof(float));
    hipMemcpy(h_fitness, dev_fitness, n_pop * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < n_pop; i++) {
        printf("fitness[%d] = %f\n", i, h_fitness[i]);
    }
    free(h_fitness);
}

void print_best_local_fitness(float* dev_best_fit, int n_pop){
    float *h_best_fitness = (float*)malloc(n_pop * sizeof(float));
    hipMemcpy(h_best_fitness, dev_best_fit, n_pop * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < n_pop; i++) {
        printf("best fitness[%d] = %f\n", i, h_best_fitness[i]);
    }
    free(h_best_fitness);
}

void print_best_position(float* dev_best_pos, int pop_size){
    float *h_best_pop = (float*)malloc(pop_size * sizeof(float));
    hipMemcpy(
        h_best_pop,
        dev_best_pos,
        pop_size * sizeof(float),
        hipMemcpyDeviceToHost
    );
    for (int i = 0; i < pop_size; i++) {
        printf("best pop[%d] = %f\n", i, h_best_pop[i]);
    }
    free(h_best_pop);
}

void print_best_global_position(float* dev_global_best_pos, int dimension){
    float *h_global_best_pos = (float*)malloc(dimension * sizeof(float));
    hipMemcpy(
        h_global_best_pos,
        dev_global_best_pos,
        dimension * sizeof(float),
        hipMemcpyDeviceToHost
    );
    for (int i = 0; i < dimension; i++) {
        printf("global best pos[%d] = %f\n", i, h_global_best_pos[i]);
    }
    free(h_global_best_pos);
}

int main(int argc, char **argv){

    float tempo_total_acumulado = 0.0f;
    float milliseconds = 0;
    int seed = SEED;
    float h_global_best_fitness;

    int dimension, n_pop, run_number, threads;
    bool verbose;

    if (argc > 1) {
        // If command line arguments are provided, use them to set parameters
        dimension = atoi(argv[1]);
        n_pop = atoi(argv[2]);
        threads = atoi(argv[3]);
        run_number = atoi(argv[4]);
        verbose = false;
    }
    else{
        dimension = DIMENSION;
        n_pop = NUMPARTICLES;
        run_number = 0;
        threads = NTHREADS;
        verbose = false;
    }

    const int pop_size = dimension * n_pop;
    int blocks = (pop_size + threads - 1) / threads;

    float *h_best_fitness = (float*)malloc(n_pop * sizeof(float));
    float *h_best_pop = (float*)malloc(pop_size * sizeof(float));


    if(verbose){
        printf("Running PSO with dimension=%d, n_pop=%d, threads=%d, blocks=%d, pop_size=%d\n", dimension, n_pop, threads, blocks, pop_size);
    }

    //memoria de constantes
    float h_lb = -5, h_ub = 5, h_IW = 0.7, h_CP = 0.5, h_SP = 0.5;
    hipMemcpyToSymbol(HIP_SYMBOL(pso_lower_bound), &h_lb, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(pso_upper_bound), &h_ub, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(pso_inertial_weight), &h_IW, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(pso_cognitive_param), &h_CP, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(pso_social_param), &h_SP, sizeof(float));

    // population, velocity
    float *dev_population, *dev_velocity, *dev_best_pos;
    hipMalloc((void**)&dev_population, pop_size * sizeof(float));
    hipMalloc((void**)&dev_velocity, pop_size * sizeof(float));
    hipMalloc((void**)&dev_best_pos, pop_size * sizeof(float));

    float *dev_global_best_pos, *dev_global_best_fitness;
    hipMalloc((void**)&dev_global_best_pos, dimension * sizeof(float));
    hipMalloc((void**)&dev_global_best_fitness, sizeof(float));

    float *dev_fitness, *dev_best_fit;
    hipMalloc((void**)&dev_fitness, n_pop * sizeof(float));
    hipMalloc((void**)&dev_best_fit, n_pop * sizeof(float));

    // inicializa velocidade e melhores posições com 0
    hipMemset(dev_velocity, 0.0f, pop_size * sizeof(float));
    hipMemset(dev_global_best_pos, 0.0f, dimension * sizeof(float));

    // inicializa fitness com zero e best individuals fitness com infinito
    float *h_fitness_init = (float*)malloc(n_pop * sizeof(float));
    float *h_best_fitness_init = (float*)malloc(n_pop * sizeof(float));
    for (int i = 0; i < n_pop; i++) {
        h_best_fitness_init[i] = INF;
        h_fitness_init[i] = 0.0f;
    }
    hipMemcpy(
        dev_best_fit,
        h_best_fitness_init,
        n_pop * sizeof(float),
        hipMemcpyHostToDevice
    );
    hipMemcpy(
        dev_fitness,
        h_fitness_init,
        n_pop * sizeof(float),
        hipMemcpyHostToDevice
    );
    free(h_fitness_init);
    free(h_best_fitness_init);

    float h_inf = INF;
    hipMemcpy(
        dev_global_best_fitness,
        &h_inf,
        sizeof(float),
        hipMemcpyHostToDevice
    );

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    init_population<<<blocks, threads>>>(
        dev_population,
        dev_best_pos,
        seed,
        n_pop,
        dimension
    );

    calc_fitness<<<blocks, threads>>>(
        dev_population,
        dev_fitness,
        n_pop,
        dimension
    );

    update_best_positions<<<blocks, threads>>>(
        dev_population,
        dev_best_pos,
        dev_fitness,
        dev_best_fit,
        n_pop,
        dimension
    );

    int fbest_blocks = (n_pop + threads - 1) / threads;
    update_best_fitness<<<fbest_blocks, threads>>>(
        dev_fitness,
        dev_best_fit,
        n_pop
    );

    float min_value = INF;
    int min_index = -1;
    hipMemcpy(
        h_best_pop,
        dev_best_pos,
        pop_size * sizeof(float),
        hipMemcpyDeviceToHost
    );
    hipMemcpy(
        h_best_fitness,
        dev_best_fit,
        n_pop * sizeof(float),
        hipMemcpyDeviceToHost
    );


    for (int i = 0; i < n_pop; i++) {
        if (h_best_fitness[i] < min_value) {
            min_value = h_best_fitness[i];
            min_index = i;
        }
    }

    hipMemcpy(
        dev_global_best_pos,
        &h_best_pop[min_index * dimension],
        dimension * sizeof(float),
        hipMemcpyHostToDevice
    );

    update_global_best<<<1, n_pop, n_pop * sizeof(float)>>>(
        dev_fitness,
        dev_global_best_fitness,
        n_pop
    );

    if (verbose){
        hipMemcpy(&h_global_best_fitness, dev_global_best_fitness, sizeof(float), hipMemcpyDeviceToHost);
        printf("Global best fitness: %f\n", h_global_best_fitness);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    tempo_total_acumulado += milliseconds;

    int generations = 100;
    int iter = 0;
    hipEventRecord(start);
    while (iter < generations) {
        update_positions<<<blocks, threads>>>(
            dev_population,
            dev_velocity,
            dev_best_pos,
            dev_global_best_pos,
            seed+iter,
            n_pop,
            dimension
        );

        hipMemset(dev_fitness, 0.0f, n_pop * sizeof(float));
        calc_fitness<<<blocks, threads, n_pop * sizeof(float)>>>(
            dev_population,
            dev_fitness,
            n_pop,
            dimension
        );

        update_best_positions<<<blocks, threads>>>(
            dev_population,
            dev_best_pos,
            dev_fitness,
            dev_best_fit,
            n_pop,
            dimension
        );

        update_best_fitness<<<fbest_blocks, threads>>>(
            dev_fitness,
            dev_best_fit,
            n_pop
        );

        if(verbose){
            print_fitness(dev_fitness, n_pop);
            print_best_local_fitness(dev_best_fit, n_pop);
            print_population(dev_population, pop_size);
        }

        update_global_best<<<1, n_pop, n_pop * sizeof(float)>>>(
            dev_fitness,
            dev_global_best_fitness,
            n_pop
        );

        hipMemcpy(
            h_best_pop,
            dev_best_pos,
            pop_size * sizeof(float),
            hipMemcpyDeviceToHost
        );
        hipMemcpy(
            h_best_fitness,
            dev_best_fit,
            n_pop * sizeof(float),
            hipMemcpyDeviceToHost
        );


        for (int i = 0; i < n_pop; i++) {
            if (h_best_fitness[i] < min_value) {
                min_value = h_best_fitness[i];
                min_index = i;
            }
        }

        hipMemcpy(
            dev_global_best_pos,
            &h_best_pop[min_index * dimension],
            dimension * sizeof(float),
            hipMemcpyHostToDevice
        );

        if(verbose && (iter % 10 == 0)){
            hipMemcpy(&h_global_best_fitness, dev_global_best_fitness, sizeof(float), hipMemcpyDeviceToHost);
            printf("\nGlobal best fitness: %f\n", h_global_best_fitness);
            print_best_global_position(dev_global_best_pos, dimension);
        }

        iter++;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    tempo_total_acumulado += milliseconds;

    hipMemcpy(&h_global_best_fitness, dev_global_best_fitness, sizeof(float), hipMemcpyDeviceToHost);
    printf("%d,%d,%d,%d,%d,%f,%.5f\n", dimension, n_pop, threads, blocks, run_number, h_global_best_fitness,tempo_total_acumulado);

    hipFree(dev_population);
    hipFree(dev_velocity);
    hipFree(dev_best_pos);
    hipFree(dev_best_fit);
    hipFree(dev_fitness);
    hipFree(dev_global_best_pos);
    hipFree(dev_global_best_fitness);

    free(h_best_fitness);
    free(h_best_pop);

    return 0;

}